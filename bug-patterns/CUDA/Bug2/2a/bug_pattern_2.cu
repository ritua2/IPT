
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>

void __global__ kernel0(int64_t Npart,int64_t* totalNpairs, int64_t* npairs){

 int64_t i =  blockIdx.x * blockDim.x + threadIdx.x;
 if(i< Npart) {
  for(int64_t j = 0;j < Npart;j++) {
      totalNpairs[(i*Npart)+j]+=7;
   }
 }
 __syncthreads();
 // the code below for reduction is incorrect and should be commented, while the commented code in the main function should be uncommented
  if(i==0){
     for(int64_t k=0; k<Npart; k++){
       for (int64_t j=0; j<Npart; j++){
              npairs[j] += totalNpairs[(k*Npart)+ j];
       }
     }
  }
 
}

//uncomment the code below for getting correct results
/*
void __global__ kernel1(int64_t *npairs, int64_t* totalNpairs, int64_t Npart){

 int64_t i =  blockIdx.x * blockDim.x + threadIdx.x;
 
 if(i==0){
     for(int64_t k=0; k<Npart; k++){
      for (int64_t j=0; j<Npart; j++){
              npairs[j] += totalNpairs[(k*Npart)+ j];
      }
     }

 }

}
*/

int main(int argc,char **argv){
  int64_t *device_npairs;
  int64_t *device_totalNpairs;

  int64_t Npart=10000;
  
  int64_t *npairs = (int64_t *) (calloc(Npart,sizeof((*npairs))));

  int64_t *totalNpairs = (int64_t *) (calloc((Npart*Npart),sizeof(( *npairs))));


  int D_rows = (Npart > 1024 ) ? Npart/1024 : Npart;
  int D_cols = (Npart > 1024 ) ? 1024 : 1;
  if ( Npart % 1024 && Npart > 1024){
    D_rows++;
  }

  printf("\nD_rows:%d, D_cols:%d\n",D_rows, D_cols);

  dim3 dimGrid(D_rows,1);
  dim3 dimBlock(D_cols,1);

  for (int64_t j=0; j<Npart; j++){
    npairs[j]  = 5;
  }
  hipMalloc((void **) &device_npairs,(Npart)*sizeof(int64_t));
  hipMemcpy(device_npairs,npairs,(Npart)*sizeof(int64_t),hipMemcpyHostToDevice);


  for (int64_t j=0; j<Npart*Npart; j++){
    totalNpairs[j]  = 0;
  }

  hipMalloc((void **) &device_totalNpairs,(Npart*Npart)*sizeof(int64_t));
  hipMemcpy(device_totalNpairs,totalNpairs,(Npart*Npart)*sizeof(int64_t),hipMemcpyHostToDevice);


  kernel0<<<dimGrid,dimBlock>>>(Npart, device_totalNpairs, device_npairs);

 
  hipMemcpy(totalNpairs,device_totalNpairs,(Npart*Npart)*sizeof(int64_t), hipMemcpyDeviceToHost); 
  //uncomment the code below to get correct results
  //kernel1<<<dimGrid,dimBlock>>>(device_npairs,device_totalNpairs, Npart);

  //cudaMemcpy(npairs,device_npairs,(Npart)*sizeof(int64_t), cudaMemcpyDeviceToHost);

  hipFree(device_npairs);
  hipFree(device_totalNpairs);

  //only printing the first 100 elements
  for (int i = 0; i < 100; i++) {
    fprintf(stdout,"%lu\t ",npairs[i]);
  }

  free(npairs);
  free(totalNpairs);
  return 0;
}
