#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void kernel0(int n, float a, float *x, float *y){

  int i = blockIdx.x*blockDim.x + threadIdx.x;
 


  //comment out this for-loop and uncomment the code in the main function for getting correct results
  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  if (i < n){
	 y[i] = a*x[i] + y[i];
  }
}

int main(){
  int N = 1048576;

  //float x[N];

  float y[N], *d_x, *d_y;

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  //for (int i = 0; i < N; i++) {
  //  x[i] = 1.0f;
  //  y[i] = 2.0f;
  //}

  //hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  //hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  kernel0<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
}
